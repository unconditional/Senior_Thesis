#include "hip/hip_runtime.h"
/* 

 Mex file for the following MATLAB function 

  function S = Szeta(zeta,k,nu4)

% Pseudospectral calculation of vorticity source term
%  S = -(- psi_y*zeta_x + psi_x*zeta_y) + nu4*del^4 zeta
% on a square periodic domain, where zeta = psi_xx + psi_yy is an NxN matrix
% of vorticity and k is vector of Fourier wavenumbers in each direction.
% Output is an NxN matrix of S at all pseudospectral gridpoints

  zetahat = fft2(zeta);
  [KX KY]  = meshgrid(k,k); % Matrix of (x,y) wavenumbers corresponding
                            % to Fourier mode (m,n)
  del2 = -(KX.^2 + KY.^2);
  del2(1,1) = 1;  % Set to nonzero to avoid division by zero when inverting
                  % Laplacian to get psi
  psihat = zetahat./del2;
  dpsidx = real(ifft2(1i*KX.*psihat));
  dpsidy = real(ifft2(1i*KY.*psihat));
  dzetadx = real(ifft2(1i*KX.*zetahat));
  dzetady = real(ifft2(1i*KY.*zetahat));
  diff4 = real(ifft2(del2.^2.*zetahat));
  S = -(-dpsidy.*dzetadx + dpsidx.*dzetady) - nu4*diff4;

  Original MATLAB code from  University of Washington
  http://www.amath.washington.edu/courses/571-winter-2006/matlab/Szeta.m

 This CUDA implementation is designed to reduce memory usage. 
 It performs the following steps:
	1) From zeta, it computes zetahat (in place)
	2) From zetahat, it computes dpsidx and dzetady (two extra arrays)
	3) It accumulates dpsidx*dzetady in S (extra array)
	4) From zetahat, it computes dpsidy and dzetadz (reuse arrays from 2)
	5) From zetahat, it computes hperviscosity (in place)
	6) It accumulates dpsidy*dzetadz and hyperviscosity in S
 

*/

#include <stdlib.h>

#include "mex.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include "hip/driver_types.h"


/* Pack real array in interleaved format  */
static __global__ void  real2complex(float *a, hipfftComplex *c, int N)
{
  unsigned int idx   = __umul24(blockIdx.x,blockDim.x)+threadIdx.x;
  unsigned int idy   = __umul24(blockIdx.y,blockDim.y)+threadIdx.y;
  if( idx<N && idy <N )
  {
  unsigned int index = idx +__umul24(idy ,N);
  c[index].x = a[index];
  c[index].y = 0.f;
  }
}


/* Compute dpsix and dzetay */
static __global__ void  der_psi_x_omega_y(hipfftComplex *c_in, 
                                          hipfftComplex *psi_x,
                                          hipfftComplex *omega_y,
                                          float *k, 
                                          int N)
{
  int idx   = __mul24(blockIdx.x,blockDim.x)+threadIdx.x;
  int idy   = __mul24(blockIdx.y,blockDim.y)+threadIdx.y;
  float scale,scale_fft;
  float2 term;
  __shared__ float kx_s[16],ky_s[16];
  if (threadIdx.y <1) kx_s[threadIdx.x] =k[idx];
  if (threadIdx.x <1) ky_s[threadIdx.y] =k[idy];
  __syncthreads();

  if( idx<N && idy <N )
  {

  int index = idx +idy *N;
  scale_fft=1.f/(N*N);
  float lkx=kx_s[threadIdx.x];
  float lky=ky_s[threadIdx.y];
  //scale=-(k[idx]*k[idx]+k[idy]*k[idy]);
  scale=-(lkx*lkx+lky*lky);
  if (idx == 0 && idy == 0) scale=1.f;

  scale =lkx/scale*scale_fft;
  term.x = c_in[index].x;
  term.y = c_in[index].y;
  psi_x[index].x =  scale*term.y;
  psi_x[index].y = -scale*term.x;
  omega_y[index].x = -lky*scale_fft*term.y;
  omega_y[index].y =  lky*scale_fft*term.x;

  }
}

/* Compute dpsiy and dzetax */
static __global__ void  der_psi_y_omega_x(hipfftComplex *c_in,
                                          hipfftComplex *psi_y,
                                          hipfftComplex *omega_x,
                                          float *k,
                                          int N)
{
  int idx   = __mul24(blockIdx.x,blockDim.x)+threadIdx.x;
  int idy   = __mul24(blockIdx.y,blockDim.y)+threadIdx.y;
  float scale, scale_fft;
  float2 term;

  __shared__ float kx_s[16],ky_s[16];
  if (threadIdx.y <1) kx_s[threadIdx.x] =k[idx];
  if (threadIdx.x <1) ky_s[threadIdx.y] =k[idy];
  __syncthreads();

  if( idx<N && idy <N )
  {
  int index = idx +idy *N;
  scale_fft=1.f/(N*N);
  float lkx=kx_s[threadIdx.x];
  float lky=ky_s[threadIdx.y];
  //scale=-(k[idx]*k[idx]+k[idy]*k[idy]);
  scale=-(lkx*lkx+lky*lky);
  if (idx == 0 && idy == 0) scale=1.f;

  //psi_y[index][0] =  scale*c_in[index][1];
  //psi_y[index][1] = -scale*c_in[index][0];
  scale =lky/scale*scale_fft;
  term.x = c_in[index].x;
  term.y = c_in[index].y;
  psi_y[index].x =  scale*term.y;
  psi_y[index].y = -scale*term.x;
  omega_x[index].x= -lkx*scale_fft*term.y;
  omega_x[index].y = lkx*scale_fft*term.x;

  }
}

/* Compute hyperviscosity term */
static __global__ void  hyperviscosity(hipfftComplex *c_in,
                                       hipfftComplex *c_out,
                                       float *k,
                                       int N)
{
  int idx   = __mul24(blockIdx.x,blockDim.x)+threadIdx.x;
  int idy   = __mul24(blockIdx.y,blockDim.y)+threadIdx.y;
  float scale,scale_fft;

  __shared__ float kx_s[16],ky_s[16];
  if (threadIdx.y <1) kx_s[threadIdx.x] =k[idx];
  if (threadIdx.x <1) ky_s[threadIdx.y] =k[idy];
  __syncthreads();

  if( idx<N && idy <N )
  {

  int index = idx +idy *N;
  scale_fft=1.f/(N*N);
  float lkx=kx_s[threadIdx.x];
  float lky=ky_s[threadIdx.y];
  scale=-(lkx*lkx+lky*lky);
  if (idx == 0 && idy == 0) scale=1.f;

  scale =scale*scale*scale_fft;
  c_out[index].x = scale *c_in[index].x;
  c_out[index].y = scale *c_in[index].y;


  }
}

/* Compute -dpsix*dzetay */
static __global__ void  non_linear_1(float *nl,
                                     hipfftComplex *a,
                                     hipfftComplex *b,
                                     int N)
{
  int idx   = __mul24(blockIdx.x,blockDim.x)+threadIdx.x;
  int idy   = __mul24(blockIdx.y,blockDim.y)+threadIdx.y;
  volatile float2 av, bv;
  if( idx<N && idy <N )
  {
  int index = idx +idy *N;
  av.x=a[index].x;
  av.y=a[index].y;
  bv.x=b[index].x;
  bv.y=b[index].y;
  nl[index] =  -av.x*bv.x;
  }
}


/* Add +dpsiy*dzetax minus hyperviscosity */
static __global__ void  non_linear_2(float *nl,
                                     hipfftComplex *a,
                                     hipfftComplex *b,
                                     hipfftComplex *c,
                                     float nu,
                                     int N)
{
  int idx   = __mul24(blockIdx.x,blockDim.x)+threadIdx.x;
  int idy   = __mul24(blockIdx.y,blockDim.y)+threadIdx.y;
  volatile float2 av, bv,cv;
  if( idx<N && idy <N )
  {
  int index = idx +idy *N;
  av.x=a[index].x;
  av.y=a[index].y;
  bv.x=b[index].x;
  bv.y=b[index].y;
  cv.x=c[index].x;
  cv.y=c[index].y;
  nl[index] +=  av.x*bv.x-nu*cv.x;
  }
}



/**************************************************************************/

/* MATLAB stores complex numbers in separate arrays for the real and
   imaginary parts.  The following functions take the data in
   this format and pack it into a complex work array, or
   unpack it, respectively.  */

void pack_r2c(hipfftComplex *input_float, 
              double *input_re, 
              int Ntot)
{
    int i;
    for (i = 0; i < Ntot; i++) 
    {
               input_float[i].x = (float) input_re[i];
               input_float[i].y = 0.0f;
    }
}

void pack_c2c(hipfftComplex *input_float, 
              double *input_re, 
              double *input_im, 
              int Ntot)
{
    int i;
    for (i = 0; i < Ntot; i++) 
    {
               input_float[i].x = (float) input_re[i];
               input_float[i].y = (float) input_im[i];
    }
}


void unpack_c2c(hipfftComplex *input_float, 
                double *output_re, 
                double *output_im,  
                int Ntot)
{
    int i;
    for (i = 0; i < Ntot; i++) 
    {
               output_re[i] = (double) input_float[i].x;
               output_im[i] = (double) input_float[i].y;
    }

}

void convert_double2float( double *input_double, float *output_float,int Ntot)
{
    int i;
    for (i = 0; i < Ntot; i++)
    {
                output_float[i] = (float) input_double[i];
    }
}

void convert_float2double( float *input_float, double *output_double,int Ntot)
{
    int i;
    for (i = 0; i < Ntot; i++)
    {
                output_double[i] = (double) input_float[i];
    }
}

/**************************************************************************/

void mexFunction( int nlhs, mxArray *plhs[],
                  int nrhs, const mxArray *prhs[])
{
  int M, N;
  double *ar, *k;
  float        *input_single ;
  float        *output_single ;
  hipfftHandle    plan;
  float  *ks, *k_d;
  double nu4;
  float nu4s;

  /* 
     Find out the  dimension of the array we want to transform:

     prhs(M,N) 
     M= Number of rows    in the mxArray prhs 
     N= Number of columns in the mxArray prhs 

  */

    if (nrhs != 3) mexPrintf ("Szeta is expecting 3 arguments\n"); 

    M = mxGetM(prhs[0]);
    N = mxGetN(prhs[0]);

    nu4 = mxGetScalar(prhs[2]);
    nu4s = (float) nu4;


  /* Allocate complex array on the device (needs to be filled 
    in interleaved format )*/
  hipfftComplex *rhs_complex_d;
  hipMalloc( (void **) &rhs_complex_d,sizeof(hipfftComplex)*N*M);

  /* Pointer for the real part of the input */
  ar =  (double *) mxGetData(prhs[0]);

 /* Compute the execution configuration */
   int block_size=16;
   dim3 dimBlock(block_size,block_size);
  
   dim3 dimGrid ( (N/dimBlock.x) + (!(N%dimBlock.x)?0:1) ,
                  (N/dimBlock.y) + (!(N%dimBlock.y)?0:1) );
 

  /* Allocating working array on host */
   input_single  = (float*) mxMalloc(sizeof(float)*N*M);

    convert_double2float(ar,input_single,  N*M); 
   /* Copy real input array to the device */
 
   float *rhs_real_d;
  hipMalloc( (void **) &rhs_real_d,sizeof(float)*N*M);

  hipMemcpy( rhs_real_d, input_single, sizeof(float)*N*M, hipMemcpyHostToDevice);

  real2complex<<<dimGrid,dimBlock>>>(rhs_real_d,rhs_complex_d,N); 

 
  /* Allocating working array on host */
    output_single  = (float *) mxMalloc(sizeof(float )*N*M);

  /* process the wave number array */
   k  = (double *) mxGetData(prhs[1]);
   ks = (float  *) mxMalloc(sizeof(float)*N);

   convert_double2float(k,ks,N);
   hipMalloc( (void **) &k_d,sizeof(float)*N);
   hipMemcpy( k_d, ks, sizeof(float)*N, hipMemcpyHostToDevice);

 


  /* Create plan for CUDA FFT 
     The current implementation is expecting a square matrix.
     In general, MATLAB is using a column-major order, CUDA a row-major order,
     so we will need to  flip the dimensions.
   */
  hipfftPlan2d(&plan, N, M, HIPFFT_C2C) ;

  /* Execute FFT on device */
  hipfftExecC2C(plan, rhs_complex_d, rhs_complex_d, HIPFFT_FORWARD) ;

  hipfftComplex *psi_d,*omega_d;
  hipMalloc( (void **) &psi_d   ,sizeof(hipfftComplex)*N*M);
  hipMalloc( (void **) &omega_d ,sizeof(hipfftComplex)*N*M);

  float *nl_d;
  hipMalloc( (void **) &nl_d   ,sizeof(float)*N*M);
 
  der_psi_x_omega_y<<<dimGrid,dimBlock>>>(rhs_complex_d,psi_d,omega_d,k_d,N);

  hipfftExecC2C(plan, psi_d, psi_d, HIPFFT_BACKWARD) ;
  hipfftExecC2C(plan, omega_d, omega_d, HIPFFT_BACKWARD) ;
  non_linear_1<<<dimGrid,dimBlock>>>(nl_d,psi_d,omega_d,N);

  der_psi_y_omega_x<<<dimGrid,dimBlock>>>(rhs_complex_d,psi_d,omega_d,k_d,N);

  hipfftExecC2C(plan, psi_d, psi_d, HIPFFT_BACKWARD) ;
  hipfftExecC2C(plan, omega_d, omega_d, HIPFFT_BACKWARD) ;

  hyperviscosity<<<dimGrid,dimBlock>>>(rhs_complex_d,rhs_complex_d,k_d,N);
  hipfftExecC2C(plan, rhs_complex_d, rhs_complex_d, HIPFFT_BACKWARD) ;
 
  non_linear_2<<<dimGrid,dimBlock>>>(nl_d,psi_d,omega_d,rhs_complex_d,nu4s,N);

  /* Destroy plan */
  hipfftDestroy(plan);

  /* Copy result back to host */
  hipMemcpy( output_single, nl_d, sizeof(float)*N*M, hipMemcpyDeviceToHost);
 
  plhs[0]=mxCreateDoubleMatrix(M,N,mxREAL);

  ar = mxGetPr(plhs[0]); 
  convert_float2double(output_single, ar, N*M); 

  /* Free the memory allocated on host and GPU */

  mxFree(input_single); mxFree(output_single); mxFree(ks);

  hipFree(k_d); hipFree(rhs_real_d); hipFree(rhs_complex_d); 
  hipFree(psi_d); hipFree(omega_d); hipFree(nl_d);

  return;
}

